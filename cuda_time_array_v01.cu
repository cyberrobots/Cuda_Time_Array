/*
 * cuda_time_array_v01.cu
 *
 *  Created on: Nov 23, 2013
 *      Author: cuda
 */



#define DEBUG

#include "libraries.h"
#include "typedefs.h"
#include "GPU_libraries.cuh"
#include "math_func.h"
#include "functions.h"
#include "my_kernels.cuh"

#define PRINT_RESULT

int main (int argc, char *argv[])
{
	int i;
	distr_var 		*array=NULL;
	distr_var 		*d_array=NULL;
	struct timeval	*Time_results=NULL;
	struct timeval	*d_Time_results=NULL;
	unsigned long	*sec;
	unsigned long	*h_sec;
	unsigned long	*ph_sec;
	unsigned long	*usec;
	unsigned long	*h_usec;
	unsigned long	*ph_usec;
	float 			run_time;
	hipEvent_t 	start, stop;
	hiprandState 	*devStates;
	hipError_t 	err = hipSuccess;
	input_var 		*input=input_check(argc,argv);
	int threadsPerBlock = 256;
	int blocksPerGrid;
	array=distribution_mix(input->globalmix,input->numofdistr,input->accurate);
	permutate(array, input->accurate);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	/*Out put Times*/
	Time_results=(struct timeval *)malloc(sizeof(struct timeval)*input->accurate);
	h_sec=(unsigned long *)malloc(sizeof(unsigned long)*input->accurate);
	h_usec=(unsigned long *)malloc(sizeof(unsigned long)*input->accurate);
	/*CPU Calculation*/
	for(i=0;i<input->accurate;i++)
	{
		delay_fix(&Time_results[i],&array[i]);
#ifdef PRINT_RESULT
		printf("%4i) sec:%1li  usec:%4li\n",i,Time_results[i].tv_sec,Time_results[i].tv_usec);
#endif
	}

	/*Pinned Memory*/
	err = hipHostAlloc((void **)&ph_sec,(size_t)(sizeof(unsigned long)*input->accurate),hipHostMallocDefault);
		if (err != hipSuccess)
		{
			fprintf(stderr,"GPU Pinned Mem.\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	err = hipHostAlloc((void **)&ph_usec,(size_t)(sizeof(unsigned long)*input->accurate),hipHostMallocDefault);
		if (err != hipSuccess)
		{
			fprintf(stderr,"GPU Pinned Mem.\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
//		/*Allocate pinned memory and initalize it.*/
//	err = hipHostAlloc((void **)&d_array,(size_t)(sizeof(distr_var)*input->accurate),hipHostMallocDefault);
//		if (err != hipSuccess)
//		{
//			fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
//			exit(EXIT_FAILURE);
//		}
//		d_array=distribution_mix(input->globalmix,input->numofdistr,input->accurate);
//		//////////////////////////////////////////
	err = hipMalloc((void **)&devStates,(size_t)(input->accurate*sizeof(hiprandState)));
		if (err != hipSuccess)
		{
			fprintf(stderr,"GPU States MEM\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	err = hipMalloc((void **)&d_array,(size_t)(sizeof(distr_var)*input->accurate));
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&sec,(size_t)(sizeof(unsigned long)*input->accurate));
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&usec,(size_t)(sizeof(unsigned long)*input->accurate));
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_array, array, (size_t)(sizeof(distr_var)*input->accurate), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU in MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	blocksPerGrid=(input->accurate + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	hipEventRecord(start, 0);
	super_kernel<<<blocksPerGrid,threadsPerBlock>>>(devStates,ph_sec,ph_usec,d_array,input->accurate);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&run_time, start, stop);
	printf ("Time for the kernel: %f ms\n", run_time);
	err = hipMemcpy(h_sec,sec,(size_t)(sizeof(unsigned long)*input->accurate),hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU out MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_usec,usec,(size_t)(sizeof(unsigned long)*input->accurate),hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU out MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


#ifdef PRINT_RESULT
	for(i=0;i<input->accurate;i++)
	{
		//printf("%4i) sec:%4li  usec:%6li\n",i,h_sec[i],h_usec[i]);
		printf("%6li,\n",ph_usec[i]);
		fflush(stderr);
	}
#endif

	hipFree(d_array);
	hipFree(d_Time_results);
	free(array);
	free(Time_results);
	free(input->globalmix);
	free(input);
	return 0;
}
