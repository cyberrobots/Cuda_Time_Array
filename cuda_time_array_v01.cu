/*
 * cuda_time_array_v01.cu
 *
 *  Created on: Nov 23, 2013
 *      Author: cuda
 */



#define DEBUG

#include "libraries.h"
#include "typedefs.h"
#include "GPU_libraries.cuh"
#include "math_func.h"
#include "functions.h"
#include "my_kernels.cuh"

#define PRINT_RESULT_CPU_NO 	// Print the calculated results

#define PRINT_RESULT_GPU_NO		// Print the calculated results

int main (int argc, char *argv[])
{
	int i;
	int threadsPerBlock 			= 	256;	// Threads per Block
	int blocksPerGrid;							// Blocks per Grid
	distr_var 		*array			=	NULL;
	distr_var 		*d_array		=	NULL;
	struct timeval	*Time_results	=	NULL;
	struct timeval	*d_Time_results	=	NULL;
	struct timeval t0;							// measuring the cpu time
	struct timeval t1;							// measuring the cpu time
	long elapsed;
	unsigned long	*sec;
	unsigned long	*h_sec;
	unsigned long	*ph_sec;
	unsigned long	*usec;
	unsigned long	*h_usec;
	unsigned long	*ph_usec;
	float 			run_time;
	hipEvent_t 	start, stop;
	hiprandState 	*devStates;
	hipError_t 	err				= 	hipSuccess;
	input_var 		*input			=	input_check(argc,argv); // grab the input from user

	/* Functions */

	// mix the array and fix the distributions according user request
	array=distribution_mix(input->globalmix,input->numofdistr,input->accurate);

	// permutate the array.
	permutate(array, input->accurate);

	// create events for time measuring
	hipEventCreate(&start);


	hipEventCreate(&stop);

	/* Out put Times */
	/* Allocate Space for the results */


	// Time structs
	Time_results=(struct timeval *)malloc(sizeof(struct timeval)*input->accurate);
	// Array for the seconds
	h_sec=(unsigned long *)malloc(sizeof(unsigned long)*input->accurate);
	// Array for the uSeconds
	h_usec=(unsigned long *)malloc(sizeof(unsigned long)*input->accurate);

	printf("Size of the calculated ensemble : %i \n",input->accurate);

	/*CPU Calculation*/
	gettimeofday(&t0,NULL);
	for(i=0;i<input->accurate;i++)
	{
		delay_fix(&Time_results[i],&array[i]);
	}
	gettimeofday(&t1,NULL);

	elapsed = (t1.tv_sec-t0.tv_sec)*1000000 + t1.tv_usec-t0.tv_usec;

	printf("The CPU processing elapsed : %f ms\n",(double)elapsed/1000);

#ifdef PRINT_RESULT_CPU
	for(i=0;i<input->accurate;i++){
		// Print the time for each
		printf("%4i) sec:%1li  usec:%4li\n",i,Time_results[i].tv_sec,Time_results[i].tv_usec);
	}
#endif

	/*Pinned Memory*/
	// Allocate
	err = hipHostAlloc((void **)&ph_sec,(size_t)(sizeof(unsigned long)*input->accurate),hipHostMallocDefault);

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU Pinned Mem.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Allocate
	err = hipHostAlloc((void **)&ph_usec,(size_t)(sizeof(unsigned long)*input->accurate),hipHostMallocDefault);

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU Pinned Mem.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
//		/*Allocate pinned memory and initalize it.*/
//	err = hipHostAlloc((void **)&d_array,(size_t)(sizeof(distr_var)*input->accurate),hipHostMallocDefault);
//		if (err != hipSuccess)
//		{
//			fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
//			exit(EXIT_FAILURE);
//		}
//		d_array=distribution_mix(input->globalmix,input->numofdistr,input->accurate);
//		//////////////////////////////////////////

	// Allocate the seed.
	err = hipMalloc((void **)&devStates,(size_t)(input->accurate*sizeof(hiprandState)));

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU States MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate data array
	err = hipMalloc((void **)&d_array,(size_t)(sizeof(distr_var)*input->accurate));

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Allocate the seconds array
	err = hipMalloc((void **)&sec,(size_t)(sizeof(unsigned long)*input->accurate));
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Allocate the useconds array
	err = hipMalloc((void **)&usec,(size_t)(sizeof(unsigned long)*input->accurate));
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU MEM\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	// Memory copy the data array
	err = hipMemcpy(d_array, array, (size_t)(sizeof(distr_var)*input->accurate), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU in MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Initialize the kernel and calculate the blocks and threads per block
	blocksPerGrid=(input->accurate + threadsPerBlock - 1) / threadsPerBlock;

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	hipEventRecord(start, 0); // start the timer.

	super_kernel<<<blocksPerGrid,threadsPerBlock>>>(devStates,ph_sec,ph_usec,d_array,input->accurate);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipEventRecord(stop, 0); // stop the timer

	hipEventSynchronize(stop); // sync the event in order to stop.

	hipEventElapsedTime(&run_time, start, stop); // calculate the time

	printf ("Time for the kernel: %f ms\n", run_time);

	// Retrieve data from the  gpu.
	// Seconds section
	err = hipMemcpy(h_sec,sec,(size_t)(sizeof(unsigned long)*input->accurate),hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU out MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// USeconds section
	err = hipMemcpy(h_usec,usec,(size_t)(sizeof(unsigned long)*input->accurate),hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr,"GPU out MEMCPY\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	/* Print the Results for debug */
#ifdef PRINT_RESULT_GPU

	for(i=0;i<input->accurate;i++)
	{
		//printf("%4i) sec:%4li  usec:%6li\n",i,h_sec[i],h_usec[i]);
		printf("%6li,\n",ph_usec[i]);
		fflush(stderr);
	}

#endif


	/* Free Memory Section */
	//	GPU
	hipFree(d_array);
	hipFree(d_Time_results);
	//	CPU
	free(array);
	free(Time_results);
	free(input->globalmix);
	free(input);

	return 0;
}
